#include "hip/hip_runtime.h"
#include "libwb/wb.h"
#include "my_timer.h"
#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

#define BLUR_SIZE 21
#define X_NTHREADS 16
#define HORIZONTAL_PASS_THREAD 32
#define Y_NTHREADS 135
#define FILTER_SIZE 1.0 / (float)((BLUR_SIZE << 1) + 1)
///////////////////////////////////////////////////////
__device__ void blur_x(int x, float *out, float *in, int width, int height) {
  // initial sum for different scenarios
  float sum = 0.0;
  // 0 <= x < Blur_size+1
  if (x < (BLUR_SIZE + 1)) {
    int padd_count = BLUR_SIZE - x;
    sum = in[0] * padd_count;
    for (int col = x; col < (BLUR_SIZE * 2 + 1 - padd_count); col++) {
      sum += in[col];
    }
    out[x] = sum * FILTER_SIZE;
  }
  // Blur_size+1 <= x < width - blur_size
  else if (x >= (BLUR_SIZE + 1) && x < (width - BLUR_SIZE)) {
    for (int col = (x - BLUR_SIZE); col < (x + BLUR_SIZE + 1); col++) {
      sum += in[col];
    }
    out[x] = sum * FILTER_SIZE;
  }
  // width-blursize <= x < width
  else if (x >= (width - BLUR_SIZE)) {
    int padd_count = x - (width - BLUR_SIZE - 1);
    sum = in[width - 1] * padd_count;
    for (int col = x; col < (BLUR_SIZE * 2 + 1 - padd_count); col++) {
      sum += in[col];
    }
    out[x] = sum * FILTER_SIZE;
  }

  for (int i = 1; i < width/HORIZONTAL_PASS_THREAD + 1; i++) {
    int col = x + i;
    if (col > width) return;
    if (col < (BLUR_SIZE + 1)) {
      sum += in[col];
      sum -= in[0];
      out[col] = sum * FILTER_SIZE;
    }
    if (col >= (BLUR_SIZE + 1) && (col < width - BLUR_SIZE)) {
      sum += in[col + BLUR_SIZE];
      sum -= in[col - BLUR_SIZE - 1];
      out[col] = sum * FILTER_SIZE;
    }
    if (col >= (width - BLUR_SIZE) && col < width) {
      sum += in[width - 1];
      sum -= in[col - BLUR_SIZE - 1];
      out[col] = sum * FILTER_SIZE;
    }
  }
}

__device__ void blur_y(float *out, float *in, int width, int height) {

  /////////////////////
  // row [0, BLUR_SIZE]
  float sum;
  sum = in[0] * BLUR_SIZE; // padding with in[0]

  // accumulate BLUR_SIZE pixels to in[0]'s top
  // and average out
  for (int row = 0; row < (BLUR_SIZE + 1); row++) {
    sum += in[row + width];
  }
  out[0] = sum * FILTER_SIZE;

  // take care of 0<row<BLUR_SIZE+1
  for (int row = 1; row < (BLUR_SIZE + 1); row++) {
    sum += in[(row + BLUR_SIZE) * width];
    sum -= in[0];
    out[row * width] = sum * FILTER_SIZE;
  }

  /////////////////////
  // row [BLUR_SIZE+1, width-BLURSIZE]
  for (int row = (BLUR_SIZE + 1); row < height - BLUR_SIZE; row++) {
    sum += in[(row + BLUR_SIZE) * width];
    sum -= in[(row - BLUR_SIZE) * width - width];
    out[row * width] = sum * FILTER_SIZE;
  }

  /////////////////////
  // row [width-BLURSIZE, width]
  for (int row = height - BLUR_SIZE; row < height; row++) {
    sum += in[(height - 1) * width]; // padding with in[h-1]
    sum -= in[(row - BLUR_SIZE) * width - width];
    out[row * width] = sum * FILTER_SIZE;
  }
}

__global__ void blurKernel_x(float *out, float *in, int width, int height) {
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int pixel_per_thread = (width)/HORIZONTAL_PASS_THREAD;// + 1;
  if (pixel_per_thread * Col < width && Row < height) {
    int start_idx_x = Col * pixel_per_thread; 
    blur_x(Col * pixel_per_thread, &out[Row * width + Col],
           &in[Row * width + Col], width, height);
  }
}

__global__ void blurKernel_y(float *out, float *in, int width, int height) {
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if (Col < width)
    blur_y(&out[Col], &in[Col], width, height);
}

void blurKernel(float *out, float *in, float *temp, int imageWidth,
                int imageHeight) {
  // horizontal pass
  {
    int nthreads = X_NTHREADS;
    dim3 threadsPerBlock(HORIZONTAL_PASS_THREAD, nthreads);
    dim3 blocksPerGrid(imageWidth/HORIZONTAL_PASS_THREAD + 1, imageHeight / nthreads);
    printf("CUDA kernel launch with [%d %d] blocks of [%d %d] threads\n",
           blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x,
           threadsPerBlock.y);
    blurKernel_x<<<blocksPerGrid, threadsPerBlock>>>(temp, in, imageWidth,
                                                     imageHeight);
  }
  // vertical pass
  {
    int nthreads = Y_NTHREADS;
    dim3 threadsPerBlock(nthreads);
    dim3 blocksPerGrid(imageWidth / nthreads );
//    printf("CUDA kernel launch with [%d %d] blocks of [%d %d] threads\n",
//           blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x,
//           threadsPerBlock.y);
    blurKernel_y<<<blocksPerGrid, threadsPerBlock>>>(out, temp, imageWidth,
                                                     imageHeight);
  }
}

///////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceTempImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  // The input image is in grayFILTER_SIZE, so the number of channels is 1
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  printf("imageWidth, imageHeight = (%d, %d)\n", imageWidth, imageHeight);
  // Since the image is monochromatic, it only contains only one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);
  
  // Use pinned memory on host
  hipHostAlloc((void **)&hostInputImageData, imageWidth * imageHeight * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void **)&hostOutputImageData, imageWidth * imageHeight * sizeof(float), hipHostMallocDefault);

  // Get host input and output image data
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  // Start timer
  timespec timer = tic();

  ///////////////////////////////////////////////////////
  // Allocate cuda memory for device input and ouput image data
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(float));

  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));

  hipMalloc((void **)&deviceTempImageData,
             imageWidth * imageHeight * sizeof(float));

  // Transfer data from CPU to GPU
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * sizeof(float), hipMemcpyHostToDevice);

  // Call your GPU kernel 10 times
  for (int i = 0; i < 1; i++) {
    blurKernel(deviceOutputImageData, deviceInputImageData, deviceTempImageData,
               imageWidth, imageHeight);
  }

  // Transfer data from GPU to CPU
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);

  // Stop and print timer
  toc(&timer, "GPU execution time (including data transfer) in seconds");

  // Check the correctness of your solution
  wbSolution(args, outputImage);

  hipHostFree(deviceInputImageData);
  hipHostFree(deviceOutputImageData);
  hipHostFree(deviceTempImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
